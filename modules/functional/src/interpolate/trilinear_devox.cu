#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "../cuda_utils.cuh"

/*
  Function: trilinear devoxlization (forward)
  Args:
    b   : batch size
    c   : #channels
    n   : number of points
    r   : voxel resolution
    r2  : r ** 2
    r3  : r ** 3
    coords : the coordinates of points, FloatTensor[b, 3, n]
    feat   : features, FloatTensor[b, c, r3]
    inds   : the voxel indices of point cube, IntTensor[b, 8, n]
    wgts   : weight for trilinear interpolation, FloatTensor[b, 8, n]
    outs   : outputs, FloatTensor[b, c, n]
*/
__global__ void trilinear_devoxelize_kernel(int b, int c, int n, int r, int r2,
                                            int r3, bool is_training,
                                            const float *__restrict__ coords,
                                            const float *__restrict__ feat,
                                            int *__restrict__ inds,
                                            float *__restrict__ wgts,
                                            float *__restrict__ outs) {
  int batch_index = blockIdx.x;
  int stride = blockDim.x;
  int index = threadIdx.x;
  coords += batch_index * n * 3;
  inds += batch_index * n * 8;
  wgts += batch_index * n * 8;
  feat += batch_index * c * r3;
  outs += batch_index * c * n;

  for (int i = index; i < n; i += stride) {
    float x = coords[i];
    float y = coords[i + n];
    float z = coords[i + n + n];
    float x_lo_f = floorf(x);
    float y_lo_f = floorf(y);
    float z_lo_f = floorf(z);

    float x_d_1 = x - x_lo_f; // / (x_hi_f - x_lo_f + 1e-8f)
    float y_d_1 = y - y_lo_f;
    float z_d_1 = z - z_lo_f;
    float x_d_0 = 1.0f - x_d_1;
    float y_d_0 = 1.0f - y_d_1;
    float z_d_0 = 1.0f - z_d_1;

    float wgt000 = x_d_0 * y_d_0 * z_d_0;
    float wgt001 = x_d_0 * y_d_0 * z_d_1;
    float wgt010 = x_d_0 * y_d_1 * z_d_0;
    float wgt011 = x_d_0 * y_d_1 * z_d_1;
    float wgt100 = x_d_1 * y_d_0 * z_d_0;
    float wgt101 = x_d_1 * y_d_0 * z_d_1;
    float wgt110 = x_d_1 * y_d_1 * z_d_0;
    float wgt111 = x_d_1 * y_d_1 * z_d_1;

    int x_lo = static_cast<int>(x_lo_f);
    int y_lo = static_cast<int>(y_lo_f);
    int z_lo = static_cast<int>(z_lo_f);
    int x_hi = (x_d_1 > 0) ? -1 : 0;
    int y_hi = (y_d_1 > 0) ? -1 : 0;
    int z_hi = (z_d_1 > 0) ? 1 : 0;

    int idx000 = x_lo * r2 + y_lo * r + z_lo;
    int idx001 = idx000 + z_hi;      // x_lo * r2 + y_lo * r + z_hi;
    int idx010 = idx000 + (y_hi & r);  // x_lo * r2 + y_hi * r + z_lo;
    int idx011 = idx010 + z_hi;      // x_lo * r2 + y_hi * r + z_hi;
    int idx100 = idx000 + (x_hi & r2); // x_hi * r2 + y_lo * r + z_lo;
    int idx101 = idx100 + z_hi;      // x_hi * r2 + y_lo * r + z_hi;
    int idx110 = idx100 + (y_hi & r);  // x_hi * r2 + y_hi * r + z_lo;
    int idx111 = idx110 + z_hi;      // x_hi * r2 + y_hi * r + z_hi;

    if (is_training) {
      wgts[i] = wgt000;
      wgts[i + n] = wgt001;
      wgts[i + n * 2] = wgt010;
      wgts[i + n * 3] = wgt011;
      wgts[i + n * 4] = wgt100;
      wgts[i + n * 5] = wgt101;
      wgts[i + n * 6] = wgt110;
      wgts[i + n * 7] = wgt111;
      inds[i] = idx000;
      inds[i + n] = idx001;
      inds[i + n * 2] = idx010;
      inds[i + n * 3] = idx011;
      inds[i + n * 4] = idx100;
      inds[i + n * 5] = idx101;
      inds[i + n * 6] = idx110;
      inds[i + n * 7] = idx111;
    }

    for (int j = 0; j < c; j++) {
      int jr3 = j * r3;
      outs[j * n + i] =
          wgt000 * feat[jr3 + idx000] + wgt001 * feat[jr3 + idx001] +
          wgt010 * feat[jr3 + idx010] + wgt011 * feat[jr3 + idx011] +
          wgt100 * feat[jr3 + idx100] + wgt101 * feat[jr3 + idx101] +
          wgt110 * feat[jr3 + idx110] + wgt111 * feat[jr3 + idx111];
    }
  }
}

/*
  Function: trilinear devoxlization (backward)
  Args:
    b   : batch size
    c   : #channels
    n   : number of points
    r3  : voxel cube size = voxel resolution ** 3
    inds   : the voxel indices of point cube, IntTensor[b, 8, n]
    wgts   : weight for trilinear interpolation, FloatTensor[b, 8, n]
    grad_y : grad outputs, FloatTensor[b, c, n]
    grad_x : grad inputs, FloatTensor[b, c, r3]
*/
__global__ void trilinear_devoxelize_grad_kernel(
    int b, int c, int n, int r3, const int *__restrict__ inds,
    const float *__restrict__ wgts, const float *__restrict__ grad_y,
    float *__restrict__ grad_x) {
  int batch_index = blockIdx.x;
  int stride = blockDim.x;
  int index = threadIdx.x;
  inds += batch_index * n * 8;
  wgts += batch_index * n * 8;
  grad_x += batch_index * c * r3;
  grad_y += batch_index * c * n;

  for (int i = index; i < n; i += stride) {
    int idx000 = inds[i];
    int idx001 = inds[i + n];
    int idx010 = inds[i + n * 2];
    int idx011 = inds[i + n * 3];
    int idx100 = inds[i + n * 4];
    int idx101 = inds[i + n * 5];
    int idx110 = inds[i + n * 6];
    int idx111 = inds[i + n * 7];
    float wgt000 = wgts[i];
    float wgt001 = wgts[i + n];
    float wgt010 = wgts[i + n * 2];
    float wgt011 = wgts[i + n * 3];
    float wgt100 = wgts[i + n * 4];
    float wgt101 = wgts[i + n * 5];
    float wgt110 = wgts[i + n * 6];
    float wgt111 = wgts[i + n * 7];

    for (int j = 0; j < c; j++) {
      int jr3 = j * r3;
      float g = grad_y[j * n + i];
      atomicAdd(grad_x + jr3 + idx000, wgt000 * g);
      atomicAdd(grad_x + jr3 + idx001, wgt001 * g);
      atomicAdd(grad_x + jr3 + idx010, wgt010 * g);
      atomicAdd(grad_x + jr3 + idx011, wgt011 * g);
      atomicAdd(grad_x + jr3 + idx100, wgt100 * g);
      atomicAdd(grad_x + jr3 + idx101, wgt101 * g);
      atomicAdd(grad_x + jr3 + idx110, wgt110 * g);
      atomicAdd(grad_x + jr3 + idx111, wgt111 * g);
    }
  }
}

void trilinear_devoxelize(int b, int c, int n, int r, int r2, int r3,
                          bool training, const float *coords, const float *feat,
                          int *inds, float *wgts, float *outs) {
  trilinear_devoxelize_kernel<<<b, optimal_num_threads(n)>>>(
      b, c, n, r, r2, r3, training, coords, feat, inds, wgts, outs);
  CUDA_CHECK_ERRORS();
}

void trilinear_devoxelize_grad(int b, int c, int n, int r3, const int *inds,
                               const float *wgts, const float *grad_y,
                               float *grad_x) {
  trilinear_devoxelize_grad_kernel<<<b, optimal_num_threads(n)>>>(
      b, c, n, r3, inds, wgts, grad_y, grad_x);
  CUDA_CHECK_ERRORS();
}
