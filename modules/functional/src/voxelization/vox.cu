#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "../cuda_utils.cuh"

/*
  Function: get how many points in each voxel grid
  Args:
    b      : batch size
    n      : number of points
    r      : voxel resolution
    r2     : = r * r
    r3     : s, voxel cube size = r ** 3
    coords : coords of each point, IntTensor[b, 3, n]
    ind    : voxel index of each point, IntTensor[b, n]
    cnt    : #points in each voxel index, IntTensor[b, s]
*/
__global__ void grid_stats_kernel(int b, int n, int r, int r2, int r3,
                                  const int *__restrict__ coords,
                                  int *__restrict__ ind, int *cnt) {
  int batch_index = blockIdx.x;
  int stride = blockDim.x;
  int index = threadIdx.x;
  coords += batch_index * n * 3;
  ind += batch_index * n;
  cnt += batch_index * r3;

  for (int i = index; i < n; i += stride) {
    // if (ind[i] == -1)
    //   continue;
    ind[i] = coords[i] * r2 + coords[i + n] * r + coords[i + n + n];
    atomicAdd(cnt + ind[i], 1);
  }
}

/*
  Function: average pool voxelization (forward)
  Args:
    b   : batch size
    c   : #channels
    n   : number of points
    s   : voxel cube size = voxel resolution ** 3
    ind : voxel index of each point, IntTensor[b, n]
    cnt : #points in each voxel index, IntTensor[b, s]
    feat: features, FloatTensor[b, c, n]
    out : outputs, FloatTensor[b, c, s]
*/
__global__ void avg_voxelize_kernel(int b, int c, int n, int s,
                                    const int *__restrict__ ind,
                                    const int *__restrict__ cnt,
                                    const float *__restrict__ feat,
                                    float *__restrict__ out) {
  int batch_index = blockIdx.x;
  int stride = blockDim.x;
  int index = threadIdx.x;
  ind += batch_index * n;
  feat += batch_index * c * n;
  out += batch_index * c * s;
  cnt += batch_index * s;
  for (int i = index; i < n; i += stride) {
    int pos = ind[i];
    // if (pos == -1)
    //   continue;
    int cur_cnt = cnt[pos];
    if (cur_cnt > 0) {
      float div_cur_cnt = 1.0 / static_cast<float>(cur_cnt);
      for (int j = 0; j < c; j++) {
        atomicAdd(out + j * s + pos, feat[j * n + i] * div_cur_cnt);
      }
    }
  }
}

/*
  Function: average pool voxelization (backward)
  Args:
    b      : batch size
    c      : #channels
    n      : number of points
    r3     : voxel cube size = voxel resolution ** 3
    ind    : voxel index of each point, IntTensor[b, n]
    cnt    : #points in each voxel index, IntTensor[b, s]
    grad_y : grad outputs, FloatTensor[b, c, s]
    grad_x : grad inputs, FloatTensor[b, c, n]
*/
__global__ void avg_voxelize_grad_kernel(int b, int c, int n, int r3,
                                         const int *__restrict__ ind,
                                         const int *__restrict__ cnt,
                                         const float *__restrict__ grad_y,
                                         float *__restrict__ grad_x) {
  int batch_index = blockIdx.x;
  int stride = blockDim.x;
  int index = threadIdx.x;
  ind += batch_index * n;
  grad_x += batch_index * c * n;
  grad_y += batch_index * c * r3;
  cnt += batch_index * r3;
  for (int i = index; i < n; i += stride) {
    int pos = ind[i];
    // if (pos == -1)
    //   continue;
    int cur_cnt = cnt[pos];
    if (cur_cnt > 0) {
      float div_cur_cnt = 1.0 / static_cast<float>(cur_cnt);
      for (int j = 0; j < c; j++) {
        atomicAdd(grad_x + j * n + i, grad_y[j * r3 + pos] * div_cur_cnt);
      }
    }
  }
}

void avg_voxelize(int b, int c, int n, int r, int r2, int r3, const int *coords,
                  const float *feat, int *ind, int *cnt, float *out) {
  grid_stats_kernel<<<b, optimal_num_threads(n)>>>(b, n, r, r2, r3, coords, ind,
                                                   cnt);
  avg_voxelize_kernel<<<b, optimal_num_threads(n)>>>(b, c, n, r3, ind, cnt,
                                                     feat, out);
  CUDA_CHECK_ERRORS();
}

void avg_voxelize_grad(int b, int c, int n, int s, const int *ind,
                       const int *cnt, const float *grad_y, float *grad_x) {
  avg_voxelize_grad_kernel<<<b, optimal_num_threads(n)>>>(b, c, n, s, ind, cnt,
                                                          grad_y, grad_x);
  CUDA_CHECK_ERRORS();
}
